#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// blockDim.x - num threads in a block, .x indicates 1D block labelling
// blockIdx.x - thread index number
// multiplying the above two variables gives start of block // then add the threadIdx.x offset for the particular thread

__global__ void saxpy_parallel(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n)  y[i] = a*x[i] + y[i];
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


int main()
{
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);

	std::cout << "device count = " << deviceCount << std::endl;

	int N = 256;
	// allocate vectors on host
	int size = N * sizeof(float);
	float* h_x = (float*)malloc(size);
	float* h_y = (float*)malloc(size);
	
	for (int i = 0;i<=N-1;i++)
	{
		h_x[i]=4;
		h_y[i]=2;
	}

	for (int i = 0;i<=N-1;i++)
	{
		std::cout << i << " " <<  h_y[i] << std::endl;
	}
	

	// allocate device memory
	float* d_x; float* d_y;

	checkCudaError(hipMalloc((void**) &d_x, size), "malloc1");
	checkCudaError(hipMalloc((void**) &d_y, size), "malloc2");

	checkCudaError(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice), "memcpy 1");
	checkCudaError(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice), "memcpy 2");

	// calculate number of blocks needed for N 
	int nblocks = (N+255)/256;

	// call 
	saxpy_parallel<<<nblocks,256>>>(N,2.0,d_x,d_y);

	checkCudaError(hipGetLastError(), "kernel launch");
	
	// Copy results back from device memory to host memory
	// implicty waits for threads to excute
	checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");
	checkCudaError(hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost), "memcpy 3");

	for (int i = 0;i<=N-1;i++)
	{
		std::cout << i << " last loop " <<  h_y[i] << std::endl;
	}



	hipFree(d_x);
	hipFree(d_y);

	free(h_x);
	free(h_y);



	return 0;

}
